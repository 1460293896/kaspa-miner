#include "hip/hip_runtime.h"
#include<stdint.h>
#include <assert.h>
#include "keccak-tiny.c"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef uint16_t MatrixRow[64];
typedef uint8_t Hash[32];
typedef uint64_t testme[64];

typedef union _uint256_t {
    uint64_t number[4];
    uint8_t hash[32];
} uint256_t;

#define BLOCKDIM 1024

#define LT_U256(X,Y) (X.number[3] != Y.number[3] ? X.number[3] < Y.number[3] : X.number[2] != Y.number[2] ? X.number[2] < Y.number[2] : X.number[1] != Y.number[1] ? X.number[1] < Y.number[1] : X.number[0] < Y.number[0])


__device__ __inline__ uint32_t amul4bit(uint32_t packed_vec1[32], uint32_t packed_vec2[32]) {
    // We assume each 32 bits have four values: A0 B0 C0 D0
    unsigned int res = 0;
    #pragma unroll
    for (int i=0; i<32; i++) {
        #if __CUDA_ARCH__ >= 610
        asm("dp4a.u32.u32" " %0, %1, %2, %3;": "=r" (res): "r" (packed_vec1[i]), "r" (packed_vec2[i]), "r" (res));
        #else
        char4 &a4 = *((char4*)&packed_vec1[i]);
        char4 &b4 = *((char4*)&packed_vec2[i]);
        res += a4.x*b4.x;
        //c += a4.y*b4.y; // In our code, the second and forth bytes are empty
        res += a4.z*b4.z;
        // c += a4.w*b4.w; // In our code, the second and forth bytes are empty
        #endif
    }

    return res;
}


extern "C" {
    //hiprandDirectionVectors64_t is uint64_t[64]
    __global__ void init(hiprandDirectionVectors64_t *seeds,  hiprandStateSobol64_t* states, const uint64_t state_count) {
        uint64_t workerId = threadIdx.x + blockIdx.x*blockDim.x;
        if (workerId < state_count) {
            hiprand_init(seeds[workerId], 0, states + workerId);
            hiprand(states + workerId);
        }
    }

    __global__ void matrix_mul(const MatrixRow *rows, const uint64_t rows_len, const Hash *hashes, const uint64_t hashes_len, Hash *outs)
    {
        int rowId = threadIdx.x + blockIdx.x*blockDim.x;
        int hashId = threadIdx.y + blockIdx.y*blockDim.y;
        //assert((rowId != 0) || (hashId != 0) );

        if (rowId < rows_len/2 && hashId < hashes_len) {
            uint16_t packed_hash[64] = {0};
            #pragma unroll
            for (int i=0; i<32; i++) {
                packed_hash[2*i] = (uint16_t)((hashes[hashId][i] & 0xF0) >> 4 );
                packed_hash[2*i+1] = (uint16_t)((hashes[hashId][i] & 0x0F));
            }
            uint32_t product1 = amul4bit((uint32_t *)(rows[(2*rowId)]), (uint32_t *)(packed_hash)) >> 10;
            uint32_t product2 = amul4bit((uint32_t *)(rows[(2*rowId+1)]), (uint32_t *)(packed_hash)) >> 10;


            outs[hashId][rowId] = hashes[hashId][rowId] ^ ((uint8_t)(product1 << 4) | (uint8_t)(product2));
            }
    }

    __global__ void pow_cshake(const uint8_t *header, uint64_t *nonces, const uint64_t nonces_len, Hash *hashes, const bool generate, hiprandStateSobol64_t* states) {
        // assuming header_len is 72
        int nonceId = threadIdx.x + blockIdx.x*blockDim.x;
        if (nonceId < nonces_len) {
            if (generate) nonces[nonceId] = hiprand(states + nonceId);
            uint8_t input[216] = {
                0x01, 0x88, // left_encode(136)                  - cSHAKE256 specific
                0x01, 0x00, // left_encode(0)                    - No Domain
                0x01, 0x78, // left_encode customization string length
                0x50, 0x72, 0x6f, 0x6f, 0x66, 0x4f, 0x66, 0x57, 0x6f, 0x72, 0x6b, 0x48, 0x61, 0x73, 0x68, // ProofOfWorkHash
            };
            // header
            memcpy(input +  136, header, 72);
            // data
            // TODO: check endianity?
            memcpy(input +  208, (uint8_t *)(nonces + nonceId), 8);
            hash(hashes[nonceId], 32, input, 216, 136, 0x04);
        }
    }

    __global__ void heavy_hash_cshake(const uint64_t *nonces, const Hash *datas, const uint64_t data_len, uint64_t *final_nonces, Hash *hashes/*, Hash *all_hashes*/) {
        assert(blockDim.x <= BLOCKDIM);
        uint64_t dataId = threadIdx.x + blockIdx.x*blockDim.x;
        if (dataId < data_len) {
            uint8_t input[168] = {
                0x01, 0x88, // left_encode(136)                  - cSHAKE256 specific
                0x01, 0x00, // left_encode(0)                    - No Domain
                0x01, 0x48, // left_encode customization string length
                0x48, 0x65, 0x61, 0x76, 0x79, 0x48, 0x61, 0x73, 0x68, //HeavyHash
                // the rest is zeros
            };
            // data
            memcpy(input +  136, datas[dataId], 32);

            __shared__ uint256_t working_hashes[BLOCKDIM]; // Shared within the block
            __shared__ uint64_t working_nonces[BLOCKDIM];

            hash(working_hashes[threadIdx.x].hash, 32, input, 168, 136, 0x04);
            working_nonces[threadIdx.x] = nonces[dataId];

            //memcpy(all_hashes + dataId, working_hashes[threadIdx.x].hash, 32);
            __syncthreads();

            // Find the minimal hash - reduce step
            for (uint64_t size = blockDim.x/2; size>0; size/=2) {
                if (threadIdx.x<size) {
                    if (
                        (working_nonces[threadIdx.x+size] != 0) &&
                        (LT_U256(working_hashes[threadIdx.x+size], working_hashes[threadIdx.x]))
                        ){
                        //memcpy(working_hashes[threadIdx.x].number, datas[dataId], 32);
                        working_hashes[threadIdx.x] = working_hashes[threadIdx.x+size];
                        working_nonces[threadIdx.x] = working_nonces[threadIdx.x+size];
                    }
                }
                __syncthreads();
            }
            if (threadIdx.x == 0) {
                final_nonces[blockIdx.x] = working_nonces[0];
                //hashes[blockIdx.x] = working_hashes[0];
                memcpy(hashes + blockIdx.x, working_hashes[0].hash, 32);
            }
        }
    }
}