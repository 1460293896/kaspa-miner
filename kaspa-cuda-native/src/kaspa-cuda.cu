#include "hip/hip_runtime.h"
#include<stdint.h>
#include <assert.h>
#include "keccak-tiny.c"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef uint8_t Hash[32];

typedef union _uint256_t {
    uint64_t number[4];
    uint8_t hash[32];
} uint256_t;

#define BLOCKDIM 1024
#define MATRIX_SIZE 64
#define HALF_MATRIX_SIZE 32
#define QUARTER_MATRIX_SIZE 16
#define HASH_HEADER_SIZE 72

#define LT_U256(X,Y) (X.number[3] != Y.number[3] ? X.number[3] < Y.number[3] : X.number[2] != Y.number[2] ? X.number[2] < Y.number[2] : X.number[1] != Y.number[1] ? X.number[1] < Y.number[1] : X.number[0] < Y.number[0])

__constant__ uint8_t matrix[MATRIX_SIZE][MATRIX_SIZE];
__constant__ uint8_t hash_header[HASH_HEADER_SIZE];
__constant__ uint256_t target;


__device__ __inline__ uint32_t amul4bit(uint32_t packed_vec1[32], uint32_t packed_vec2[32]) {
    // We assume each 32 bits have four values: A0 B0 C0 D0
    unsigned int res = 0;
    #pragma unroll
    for (int i=0; i<QUARTER_MATRIX_SIZE; i++) {
        #if __CUDA_ARCH__ >= 610
        asm("dp4a.u32.u32" " %0, %1, %2, %3;": "=r" (res): "r" (packed_vec1[i]), "r" (packed_vec2[i]), "r" (res));
        #else
        char4 &a4 = *((char4*)&packed_vec1[i]);
        char4 &b4 = *((char4*)&packed_vec2[i]);
        res += a4.x*b4.x;
        res += a4.y*b4.y; // In our code, the second and forth bytes are empty
        res += a4.z*b4.z;
        res += a4.w*b4.w; // In our code, the second and forth bytes are empty
        #endif
    }

    return res;
}


extern "C" {
    //hiprandDirectionVectors64_t is uint64_t[64]
    __global__ void init(hiprandDirectionVectors64_t *seeds,  hiprandStateSobol64_t* states, const uint64_t state_count) {
        uint64_t workerId = threadIdx.x + blockIdx.x*blockDim.x;
        if (workerId < state_count) {
            hiprand_init(seeds[workerId], 0, states + workerId);
            hiprand(states + workerId);
        }
    }

    __global__ void matrix_mul(const Hash *hashes, const uint64_t hashes_len, Hash *outs)
    {
        int rowId = threadIdx.x + blockIdx.x*blockDim.x;
        int hashId = threadIdx.y + blockIdx.y*blockDim.y;
        //assert((rowId != 0) || (hashId != 0) );

        if (rowId < HALF_MATRIX_SIZE && hashId < hashes_len) {
            uchar4 packed_hash[QUARTER_MATRIX_SIZE] = {0};
            #pragma unroll
            for (int i=0; i<QUARTER_MATRIX_SIZE; i++) {
                packed_hash[i] = make_uchar4(
                    (hashes[hashId][2*i] & 0xF0) >> 4 ,
                    (hashes[hashId][2*i] & 0x0F),
                    (hashes[hashId][2*i+1] & 0xF0) >> 4,
                    (hashes[hashId][2*i+1] & 0x0F)
                );
            }
            uint32_t product1 = amul4bit((uint32_t *)(matrix[(2*rowId)]), (uint32_t *)(packed_hash)) >> 10;
            uint32_t product2 = amul4bit((uint32_t *)(matrix[(2*rowId+1)]), (uint32_t *)(packed_hash)) >> 10;


            outs[hashId][rowId] = hashes[hashId][rowId] ^ ((uint8_t)(product1 << 4) | (uint8_t)(product2));
            }
    }

    __global__ void pow_cshake(uint64_t *nonces, const uint64_t nonces_len, Hash *hashes, const bool generate, hiprandStateSobol64_t* states) {
        // assuming header_len is 72
        int nonceId = threadIdx.x + blockIdx.x*blockDim.x;
        if (nonceId < nonces_len) {
            if (generate) nonces[nonceId] = hiprand(states + nonceId);
            uint8_t input[216] = {
                0x01, 0x88, // left_encode(136)                  - cSHAKE256 specific
                0x01, 0x00, // left_encode(0)                    - No Domain
                0x01, 0x78, // left_encode customization string length
                0x50, 0x72, 0x6f, 0x6f, 0x66, 0x4f, 0x66, 0x57, 0x6f, 0x72, 0x6b, 0x48, 0x61, 0x73, 0x68, // ProofOfWorkHash
            };
            // header
            memcpy(input +  136, hash_header, HASH_HEADER_SIZE);
            // data
            // TODO: check endianity?
            memcpy(input +  208, (uint8_t *)(nonces + nonceId), 8);
            hash(hashes[nonceId], 32, input, 216, 136, 0x04);
        }
    }

    __global__ void heavy_hash_cshake(const uint64_t *nonces, const Hash *datas, const uint64_t data_len, uint64_t *final_nonce/*, Hash *all_hashes*/) {
        assert(blockDim.x <= BLOCKDIM);
        uint64_t dataId = threadIdx.x + blockIdx.x*blockDim.x;
        if (dataId < data_len) {
            uint8_t input[168] = {
                0x01, 0x88, // left_encode(136)                  - cSHAKE256 specific
                0x01, 0x00, // left_encode(0)                    - No Domain
                0x01, 0x48, // left_encode customization string length
                0x48, 0x65, 0x61, 0x76, 0x79, 0x48, 0x61, 0x73, 0x68, //HeavyHash
                // the rest is zeros
            };
            // data
            memcpy(input +  136, datas[dataId], 32);

            uint256_t working_hash;
            hash(working_hash.hash, 32, input, 168, 136, 0x04);
            if (LT_U256(working_hash, target)){
                atomicCAS((unsigned long long int*) final_nonce, 0, (unsigned long long int) nonces[dataId]);
            }
        }
    }
}